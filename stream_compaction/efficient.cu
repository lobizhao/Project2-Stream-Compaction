#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        //up sweep
        __global__ void upSweep(int n, int d, int *data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;
            if (k < n && k % (1 << (d + 1)) == 0) {
                data[k + (1<<(d+1)) - 1] += data[k + (1<<d) - 1];
            }
        }

        //down sweep
        __global__ void downSweep(int n, int d, int *data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;
            if (k < n && k % (1 << (d + 1)) == 0) {
                int temp = data[k + (1 << d) - 1];
                data[k + (1 << d) - 1] = data[k + (1 << (d + 1)) - 1];
                data[k + (1 << (d + 1)) - 1] += temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            int pow2n = 1 << ilog2ceil(n);
            int *dev_data;
            hipMalloc((void**)&dev_data, pow2n * sizeof(int));
            hipMemset(dev_data, 0, pow2n * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            dim3 blockSize(128);
            dim3 gridSize((pow2n + blockSize.x - 1) / blockSize.x);
            
            //run Up sweep
            for (int d = 0; d < ilog2ceil(pow2n); d++) {
                upSweep<<<gridSize, blockSize>>>(pow2n, d, dev_data);
            }
            
            hipMemset(&dev_data[pow2n - 1], 0, sizeof(int));
            
            //run Down sweep
            for (int d = ilog2ceil(pow2n) - 1; d >= 0; d--) {
                downSweep<<<gridSize, blockSize>>>(pow2n, d, dev_data);
            }
            
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_data);
            
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_bools, n * sizeof(int));
            hipMalloc((void**)&dev_indices, n * sizeof(int));
            
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            dim3 blockSize(128);
            dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
            
            //map to boolean - b array 
            StreamCompaction::Common::kernMapToBoolean<<<gridSize, blockSize>>>(n, dev_bools, dev_idata);
            
            //exclusive scan on boolean array
            int *tempBools = new int[n];
            hipMemcpy(tempBools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
            int *tempIndices = new int[n];
            scan(n, tempIndices, tempBools);
            hipMemcpy(dev_indices, tempIndices, n * sizeof(int), hipMemcpyHostToDevice);

            //scatter
            StreamCompaction::Common::kernScatter<<<gridSize, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
            
            int finalBool;
            int finalIndex;
            hipMemcpy(&finalBool, &dev_bools[n-1], sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&finalIndex, &dev_indices[n-1], sizeof(int), hipMemcpyDeviceToHost);
            int count = finalIndex + finalBool;
            
            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);

            delete[] tempBools;
            delete[] tempIndices;
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            
            timer().endGpuTimer();
            return count;
        }
    }
}
