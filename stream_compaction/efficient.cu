#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        //up sweep
        __global__ void upSweep(int n, int d, int *data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;
            if (k < n && k % (1 << (d + 1)) == 0) {
                data[k + (1<<(d+1)) - 1] += data[k + (1<<d) - 1];
            }
        }

        //down sweep
        __global__ void downSweep(int n, int d, int *data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;
            if (k < n && k % (1 << (d + 1)) == 0) {
                int temp = data[k + (1 << d) - 1];
                data[k + (1 << d) - 1] = data[k + (1 << (d + 1)) - 1];
                data[k + (1 << (d + 1)) - 1] += temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            int log2n = 1 << ilog2ceil(n);
            int *dev_data;
            hipMalloc((void**)&dev_data, log2n * sizeof(int));
            hipMemset(dev_data, 0, log2n * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            dim3 blockSize(128);
            dim3 gridSize((log2n + blockSize.x - 1) / blockSize.x);
            
            //run Up sweep
            for (int d = 0; d < ilog2ceil(log2n); d++) {
                upSweep<<<gridSize, blockSize>>>(pow2n, d, dev_data);
            }
            
            hipMemset(&dev_data[log2n - 1], 0, sizeof(int));
            
            //run Down sweep
            for (int d = ilog2ceil(log2n) - 1; d >= 0; d--) {
                downSweep<<<gridSize, blockSize>>>(pow2n, d, dev_data);
            }
            
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_data);
            
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
