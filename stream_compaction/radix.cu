#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "radix.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Radix {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // only get the 0th binary 
        __global__ void get0thBit(int n, int *bits, const int *data, int bitPos) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            bits[index] = (data[index] >> bitPos) & 1;
        }

        //get inverse array set 0th bit as 1
        __global__ void invert0thBits(int n, int *notBits, const int *bits) {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            notBits[index] = 1 - bits[index];
        }


        __global__ void radixScatter(int n, int *odata, const int *idata, 
                                       const int *bits, const int *falseIndices, 
                                       const int *trueIndices, int numFalse) {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) {
                return;
            }
            
            if (bits[index] == 0) {
                odata[falseIndices[index]] = idata[index];
            } else {
                odata[numFalse + trueIndices[index]] = idata[index];
            }
        }


        void sort(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            int* dev_input;
            int* dev_output;
            int* dev_bits;
            int* dev_notBits;
            int* dev_falseIndices;
            int* dev_trueIndices;
            
            hipMalloc((void**)&dev_input, n * sizeof(int));
            hipMalloc((void**)&dev_output, n * sizeof(int));
            hipMalloc((void**)&dev_bits, n * sizeof(int));
            hipMalloc((void**)&dev_notBits, n * sizeof(int));
            hipMalloc((void**)&dev_falseIndices, n * sizeof(int));
            hipMalloc((void**)&dev_trueIndices, n * sizeof(int));
            

            hipMemcpy(dev_input, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            dim3 blockSize(BLOCK_SIZE);
            dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
            

            for (int bit = 0; bit < 32; bit++) {
                //get the 0th binary bit
                get0thBit <<<gridSize, blockSize>>>(n, dev_bits, dev_input, bit);
                
                //invert array b 
                invert0thBits <<<gridSize, blockSize>>>(n, dev_notBits, dev_bits);
                
                StreamCompaction::Efficient::scanOnGpu(n, dev_falseIndices, dev_notBits);             
                StreamCompaction::Efficient::scanOnGpu(n, dev_trueIndices, dev_bits);
                
                int lastNotBit;
                int lastFalseIndex;

                hipMemcpy(&lastNotBit, &dev_notBits[n-1], sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastFalseIndex, &dev_falseIndices[n-1], sizeof(int), hipMemcpyDeviceToHost);

                //sum offset 
                int numFalse = lastFalseIndex + lastNotBit;
                radixScatter <<<gridSize, blockSize>>>(n, dev_output, dev_input,
                                                        dev_bits, dev_falseIndices, 
                                                        dev_trueIndices, numFalse);
                
                int *temp = dev_input;
                dev_input = dev_output;
                dev_output = temp;
            }
            
            hipMemcpy(odata, dev_input, n * sizeof(int), hipMemcpyDeviceToHost);
            //dele

            hipFree(dev_input);
            hipFree(dev_output);
            hipFree(dev_bits);
            hipFree(dev_notBits);
            hipFree(dev_falseIndices);
            hipFree(dev_trueIndices);
            
            timer().endGpuTimer();
        }
    }
}